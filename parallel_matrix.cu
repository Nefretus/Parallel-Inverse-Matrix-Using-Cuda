#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <random>

#define N 512

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N*sizeof(int);
	// Allocate space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	// Setup input values
	a = (int*)malloc(size);
	b = 7;
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU
	add <<<N, 1>>> (d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}